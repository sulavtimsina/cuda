/*
Submitted By: Sulav Timsina
ID: 50502493
Course: CS 6253  Heterogeneous Computing
Spring , 2018
Submitted On; 04/16/2018
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>


/*   Array to print an array to Console
    arr = pointer to array
    dimen = 1-D size of array
*/
void printArray(int *arr, int dimen){
    //printing vectors
    int i;
    for ( i = 0; i < dimen*dimen; i++){
      printf("%d ",arr[i]);
      if((i+1) % dimen == 0)
        printf("\n");    
    }
}

/*   CUDA Kernel function to add the elements of two arrays on the GPU
     n = 1-D size of the array
     a, b = 
*/
__global__ void kernel1(int n, int *a, int *b, int *c){
  
  int index = threadIdx.x;
  int stride = blockDim.x;
  
  for( int i = index; i < n*n; i += stride){
    c[i] = a[i] + b[i];
  }
}

__global__ void kernel2(int n, int *a, int *b, int *c){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for( int i = index; i < n*n; i += stride){
    c[i] = a[i] + b[i];
  }
}

__global__ void kernel3(int n, int *a, int *b, int *c){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for( int i = index; i < n*n; i += stride){
    c[i] = a[i] + b[i];
  }
}



int main(int argc, char* argv[] ){
  //size of vector
  int n = 8;
  int N = n * n;  
  
  // Host input vectors
  int *h_a, *h_b;
  // Host output vectors
  int *h_c;
  // Device input vectors
  int *d_a, *d_b;
  // Device output vectors
  int *d_c;
  
  // Allocate memory for each vector of host
  h_a = (int*)malloc(N * sizeof(int));
  h_b = (int*)malloc(N * sizeof(int));
  h_c = (int*)malloc(N * sizeof(int));

  // Allocate memory for each vector of device
  hipMalloc( &d_a, N * sizeof(int) );
  hipMalloc( &d_b, N * sizeof(int) );
  hipMalloc( &d_c, N * sizeof(int) );
  
  // Initialize vectors on host
  int i, j;
  for ( i = 0; i < n; i++){
    for ( j = 0; j < n; j++){
      h_a[i*n + j] = i + j;
      h_b[i*n + j] = i + j;
    }  
  }
  
  // Print Vector a
  printf("Vector a:\n");
  printArray(h_a, n); 
  // Print Vector b
  printf("\nVector b:\n");
  printArray(h_b, n);
   
  // Copy host vectors to device
  hipMemcpy( d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy( d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
  
  /****************************************************************************/
  // Run the kernel1 function on the GPU
  kernel1<<<1,256>>>(n, d_a, d_b, d_c);
    
  // Copy array C back to host memory
  hipMemcpy( h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost );
  
  //Print Vector c
  printf("\nVector c(Output of kernel1):\n");
  printArray(h_c, n);
  /****************************************************************************/
  
  //kernel 2 call
  int blockSize = 16;
  int numBlocks = 4;
  kernel2<<<numBlocks, blockSize>>>(n, d_a, d_b, d_c);
  
  // Copy array C back to host memory
  hipMemcpy( h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost );
  
  //Print Vector c
  printf("\nVector c(Output of kernel2):\n");
  printArray(h_c, n);
  /****************************************************************************/
  //kernel 3 call
  blockSize = 4;
  numBlocks = 16;
  kernel3<<<numBlocks, blockSize>>>(n, d_a, d_b, d_c);  
  
  // Copy array C back to host memory
  hipMemcpy( h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost );
  
  //Print Vector c
   printf("\nVector c(Output of kernel3):\n");
  printArray(h_c, n);
  
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;  
}