/*
Submitted By: Sulav Timsina
ID: 50502493
Course: CS 6253  Heterogeneous Computing
Spring , 2018
Submitted On; 04/16/2018
*/
/*
The device property can also be found from command line using the command:
lshw -C display
*/
#include <stdio.h> 

int main() {
  int nDevices;
  
  hipGetDeviceCount(&nDevices);
  
  printf("Number of GPUs %d\n",nDevices);
  printf("***************************\n***************************\n");
  
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    /*prop is a structure which contains different properties of processors as its element*/
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
           
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
           
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
           
    printf("  Multiprocessor Count: %d\n",
           prop.multiProcessorCount);
           
    printf("  Maximum no. of register available to a thread block: %d\n",
           prop.regsPerBlock);
           
    printf("  Maximum no. of threads per block: %d\n",
           prop.maxThreadsPerBlock);
           
    printf("  Concurrent Kernels%d\n",
           prop.concurrentKernels);
           
    if(prop.integrated)
      printf("The device is integrated in the motherboard\n");
    else
      printf("The device is NOT integrated in the motherboard\n");
    printf("***************************\n***************************\n");
  }
}